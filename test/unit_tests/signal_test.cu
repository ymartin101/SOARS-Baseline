#include "hip/hip_runtime.h"
/// signal_test.cpp - Test program for the signal class
/// Marc Brooker, 25 May 2006
/// Edited by Yaaseen Martin, 28 August 2019

#include <iostream>
#include <math.h>
#include "rssignal.cuh"
#include "fftwcpp.h"

using namespace rsSignal;
using namespace std;

void dump(Signal &sig)
{
    cDbl *data = sig.DataPtr();
    for (int i = 0; i < sig.Size(); i++)
        cout << data[i] << "\n";
    cout << endl;
}

void fillsignal(Signal &sig, double freq, double sr, double time)
{
    int size = sr*time;
    double *data = new double[size];

    // Fill the data with a cos wave
    for (int i = 0; i < size; i++)
        data[i] = cos(i/sr*2*M_PI*freq);

    // Load the signal
    sig.Load(data, size, sr);
    delete[] data;
}


int main()
{
    Signal sig1, sig2;
    fillsignal(sig1, 1e3, 3e3, 1e-1);
    fillsignal(sig2, 1e3, 3e3, 1e-1);

    sig1.Decimate(3);
    sig1.Interpolate(3);
    sig2 -= sig1;

    FFTManager *manager = FFTManager::Instance();
    manager->Clean();
}
