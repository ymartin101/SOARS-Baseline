/// path_test.cpp - Test path implementation
/// Marc Brooker
/// Edited by Yaaseen Martin, 28 August 2019

#include <iostream>
#include "rspath.cuh"
#include "cycle.h"

using namespace std;
using namespace rs;

int main() {
    ticks t1, t2;
    Path testpath(Path::RS_INTERP_CUBIC);
    Coord coord = {0, 0, 0, 0};

    // Add coordinates
    testpath.AddCoord(coord);
    coord.x = 1; coord.t = 1;
    testpath.AddCoord(coord);
    coord.x = 3; coord.t = 2;
    testpath.AddCoord(coord);
    coord.x = -1; coord.t = 3;
    testpath.AddCoord(coord);

    t1 = getticks();
    testpath.Finalize();
    t2 = getticks();
    cout << "Setup took " << elapsed(t2, t1) << " ticks" << endl;

    t1 = getticks();
    for (int i = -100; i < 3100; i++)
        testpath.GetPosition(i/1000.0, coord);
    t2 = getticks();
    cout << "Interp took " << elapsed(t2, t1) << " ticks" << endl;

    for (int i = 0; i <= 30; i+=2) {
        testpath.GetPosition(i/10.0, coord);
        cout << coord.t << " " << coord.x << " " << coord.y << " " << coord.z << endl;
    }
    return 0;
}
