#include "hip/hip_runtime.h"
/// rsresponse.cpp - Implementation of ResponseBase and derived classes
/// Marc Brooker, 3 August 2006
/// Edited by Yaaseen Martin, 27 August 2019

#include <sstream>
#include <fstream>
#include <iomanip>
#include "rsresponse.cuh"
#include "rsradar.cuh"

/// Attach a text node to an XML element for creating structures like this:
// <node>
//   <name>text</name>
// </node>

using namespace rs;

namespace{

    // Attach a text node to an XML element
    void AttachTextNode(pugi::xml_node &root, std::string name, std::string text)
    {
        pugi::xml_node element = root.append_child(name.c_str());
        element.append_child(pugi::node_pcdata).set_value(text.c_str());
    }

    // Attach a text node to an XML element, getting the text by converting a rsFloat to a string
    void AttachRsFloatNode(pugi::xml_node &root, std::string name, rsFloat data, bool scientific = true, int precision = 10)
    {
        std::ostringstream oss;
        if (scientific)
            oss.setf(std::ios::scientific);
        oss << std::setprecision(precision) << data;
        AttachTextNode(root, name, oss.str());
    }

    // Attach a text node to an XML element, getting the text by converting an int to a string
    void AttachIntNode(pugi::xml_node &root, std::string name, int data)
    {
        std::ostringstream oss;
        oss << data;
        AttachTextNode(root, name, oss.str());
    }

}

/// Interppoint Implementation

// InterpPoint Constructor
InterpPoint::InterpPoint(rsFloat power, rsFloat start, rsFloat delay, rsFloat doppler, rsFloat phase, rsFloat noise_temperature):
    power(power),
    time(start),
    delay(delay),
    doppler(doppler),
    phase(phase),
    noise_temperature(noise_temperature)
{
}

/// ResponseBase Implementation

Response::Response(RadarSignal* wave, const Transmitter* transmitter):
    transmitter(transmitter),
    wave(wave)
{
}

Response::~Response()
{
}

// Return the time the pulse's energy starts
rsFloat Response::StartTime() const
{
    if (points.empty())
        return 0;
    return points.front().time;
}

// Return the time the pulse's energy ends
rsFloat Response::EndTime() const
{
    if (points.empty())
        return 0;
    return points.back().time;
}

// Return the length of the pulse
rsFloat Response::GetLength() const
{
    return EndTime()-StartTime();
}

// Get the name of the transmitter which caused this response
std::string Response::GetTransmitterName() const
{
    return transmitter->GetName();
}

// Return a pointer to the waveform
const rs::RadarSignal* Response::GetWave() const
{
    return wave;
}

// Render a single response point to XML
void Response::RenderResponseXML(pugi::xml_node &resp, const InterpPoint &point)
{
    // Create a node for the response
    pugi::xml_node element = resp.append_child("InterpolationPoint");

    // Attach nodes for properties of the response
    AttachRsFloatNode(element, "time", point.time, false);
    AttachRsFloatNode(element, "delay", point.delay, false);
    // AttachRsFloatNode(element, "amplitude", std::sqrt(point.power*wave->GetPower()), false);
    AttachRsFloatNode(element, "phase", point.phase, false);
    AttachRsFloatNode(element, "doppler", point.doppler, false); // Fd = (Fr - Ft), i.e. +V == +Fd target moves towards radar
    AttachRsFloatNode(element, "power", point.power*wave->GetPower());
    // AttachRsFloatNode(element, "Iamplitude", std::cos(point.phase)*std::sqrt(point.power*wave->GetPower()));
    // AttachRsFloatNode(element, "Qamplitude", std::sin(point.phase)*std::sqrt(point.power*wave->GetPower()));
    // AttachRsFloatNode(element, "noise_temperature", point.noise_temperature);
    // AttachRsFloatNode(element, "phasedeg", point.phase/M_PI*180);
}

// Render the response to an XML file
void Response::RenderXML(pugi::xml_node &rec)
{
    // Create a node for the response
    pugi::xml_node element = rec.append_child("Response");
    element.append_attribute("transmitter") = GetTransmitterName().c_str();

    // Attach nodes for properties of the response
    ::AttachRsFloatNode(element, "start", StartTime(), false);
    AttachTextNode(element, "name", wave->GetName());

    // Render each interpolation point in turn
    std::vector<InterpPoint>::iterator i;
    for (i = points.begin(); i != points.end(); i++)
        RenderResponseXML(element, *i);
}

// Render a InterpPoint as CSV
void Response::RenderResponseCSV(std::ofstream &of, const InterpPoint &point)
{
    of << point.time << ", " << point.power << ", " << point.phase << ", " << point.doppler << "\n";
}

// Render the response to a CSV file
void Response::RenderCSV(std::ofstream &of)
{
    //Render each interpolation point
    std::vector<InterpPoint>::const_iterator i;
    for (i = points.begin(); i != points.end(); i++)
        RenderResponseCSV(of, *i);
}

// Add an interp point to the vector
void Response::AddInterpPoint(InterpPoint &point)
{
    // Check that points are being added in order
    if ((!points.empty()) && (point.time < points.back().time))
        throw std::logic_error("BUG: Interpolation points not being added in order");

    // This method does not need a mutex as only one thread owns any non-const Response object
    points.push_back(point);
}

// Return the number of Points
int Response::CountPoints() const
{
    return points.size();
}

// Render the response to an array
boost::shared_array<rsComplex> Response::RenderBinary(rsFloat& rate, unsigned int &size, rsFloat frac_win_delay)
{
    rate = wave->GetRate();
    return wave->Render(points, size, frac_win_delay);
}
