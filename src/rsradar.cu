#include "hip/hip_runtime.h"
/// rsradar.cpp - Implementation of classes defined in rsradar.h
/// Marc Brooker, 21 April 2006
/// Edited by Yaaseen Martin, 02 September 2019

#include <algorithm>
#include <stdexcept>
#include <cmath>
#include <limits>
#include "rsradar.cuh"
#include "rsdebug.cuh"
#include "rspulserender.cuh"
#include "rsresponse.cuh"
#include "rsantenna.cuh"
#include "rsparameters.cuh"
#include "rspath.cuh"
#include "rstiming.cuh"
#include "rsmultipath.cuh"
#include "rsplatform.cuh"

using namespace rs; //Import the rs namespace for clarity

/// Radar Implementation

// Default Constructor
Radar::Radar(const Platform *platform, const std::string& name):
    Object(platform, name),
    timing(0),
    antenna(0),
    attached(0),
    multipath_dual(false),
    multipath_reflect(0)
{
}

// Default Destructor
Radar::~Radar()
{
}

// Attach a receiver to the transmitter for a monostatic configuration
void Radar::MakeMonostatic(Radar* recv)
{
    if (attached)
        throw std::runtime_error("BUG: Attempted to attach second receiver to transmitter");
    attached = recv;
}

// Get the attached receiver; likely to be 0 (NULL) - which means the transmitter does not share its antenna
const Radar* Radar::GetAttached() const
{
    return attached;
}

// Return whether the radar is monostatic
bool Radar::IsMonostatic() const
{
    return attached;
}

// Set the transmitter's antenna
void Radar::SetAntenna(Antenna* ant)
{
    if (!ant)
        throw std::logic_error("BUG: Transmitter's antenna set to null");
    antenna = ant;
}

// Return the antenna gain in the specified direction
rsFloat Radar::GetGain(const SVec3 &angle, const SVec3 &refangle, rsFloat wavelength) const
{
    return antenna->GetGain(angle, refangle, wavelength);
}

// Get the noise temperature (including antenna noise temperature)
rsFloat Radar::GetNoiseTemperature(const SVec3 &angle) const
{
    return antenna->GetNoiseTemperature(angle);
}

// Attach a timing object to the radar
void Radar::SetTiming(Timing* tim) {
  if (!tim)
    throw std::runtime_error("BUG: Radar timing source must not be set to NULL");
  timing = tim;
}

// Get a pointer to the radar's timing object
Timing* Radar::GetTiming() const
{
    if (!timing)
        throw std::runtime_error("BUG: Radar::GetTiming called before timing set");
    return timing;
}

// Check if object is a virtual multipath dual
bool Radar::IsMultipathDual() const
{
    return multipath_dual;
}

// Set object as a virtual multipath dual
void Radar::SetMultipathDual(rsFloat reflect)
{
    multipath_dual = true;
    multipath_reflect = reflect;

    // Sanity check the reflectance factor
    if (multipath_reflect > 1)
        rsDebug::printf(rsDebug::RS_CRITICAL, "Multipath reflection factor greater than 1 (=%g) for radar %s, results are likely to be incorrect\n", reflect, GetName().c_str());
}

// Get the reflecting factor
rsFloat Radar::MultipathDualFactor() const
{
    return multipath_reflect;
}

/// Transmitter Implementation

// Default constructor for Transmitter
Transmitter::Transmitter(const Platform *platform, const std::string& name, bool pulsed):
    Radar(platform, name),
    signal(0),
    pulsed(pulsed),
    dual(0)
{
}

// Default destructor for Transmitter
Transmitter::~Transmitter()
{
    delete GetTiming();
}

// Set the transmitter's pulse waveform
void Transmitter::SetWave(RadarSignal *wave)
{
    signal = wave;
}

// Return the number of pulses this transmitter produces over the simulation lifetime
int Transmitter::GetPulseCount() const
{
    rsFloat time = rsParameters::end_time() - rsParameters::start_time();
    if (pulsed) {
        rsFloat pulses = time*prf;
        return static_cast<int>(std::ceil(pulses));
    }
    else
        return 1; //CW systems only have one 'pulse'
}

// Fill the structure with the number'th pulse in the transmitter's pulse list
void Transmitter::GetPulse(TransmitterPulse *pulse, int number) const
{
    // Pulse waveform is same as transmitter waveform
    pulse->wave = signal;

    // Calculate start time of pulse
    if (pulsed)
        pulse->time = static_cast<rsFloat>(number)/prf; // Pulse mode start depends on PRF
    else
        pulse->time = 0; // CW transmitters start at zero for now

    // If there is timing jitter, add it
    if (!timing)
        throw std::logic_error("BUG: Transmitter "+GetName()+" must be associated with timing source");
}

// Set the Pulse Repetition Frequency of the transmitter
void Transmitter::SetPRF(rsFloat mprf)
{
    rsFloat rate = rsParameters::rate()*rsParameters::oversample_ratio();

    // The PRF must be rounded to an even number of samples
    prf = 1/(std::floor(rate/mprf)/rate);
}

/// Receiver Implementation

// Default constructor for Receiver
Receiver::Receiver(const Platform *platform, std::string name):
    Radar(platform, name),
    noise_temperature(0),
    dual(0),
    flags(0)
{
}

// Default destructor for Receiver
Receiver::~Receiver()
{
    ClearResponses();
    delete timing; // The timing is unique to the receiver
}

// Add a response to the list of responses for this receiver
void Receiver::AddResponse(Response *response)
{
    boost::try_mutex::scoped_lock lock(responses_mutex);
    responses.push_back(response);
}

// Clear the list of system responses
void Receiver::ClearResponses()
{
    std::vector<Response *>::iterator i;
    for (i = responses.begin(); i != responses.end(); i++)
        delete *i;
    responses.clear();
}

// Comparison function for response*
inline bool CompareTimes(const Response *a, const Response *b)
{
    return (a->StartTime())<(b->StartTime());
}

// Render the antenna's responses to XML/CSV file(s)
void Receiver::Render()
{
    try {
        // This mutex should never be locked, enforce that condition
        boost::try_mutex::scoped_try_lock lock(responses_mutex);

        // Sort the returns into time order
        std::sort(responses.begin(), responses.end(), CompareTimes);

        // Export the pulse descriptions to XML
        if (rsParameters::export_xml())
            ExportReceiverXML(responses, GetName() + "_results");

        // Export to CSV format
        if (rsParameters::export_csv())
            ExportReceiverCSV(responses, GetName()+"_results");

        // Unlock the mutex
        lock.unlock();
    }
    catch (boost::lock_error &e)
    {
        throw std::runtime_error("BUG: Responses lock is locked during Render()");
    }
}

// Render the antenna's responses to HDF5 file
void Receiver::RenderBin()
{
    // Export a binary containing the pulses
    if (rsParameters::export_binary())
        ExportReceiverBinary(responses, this, GetName());
}

// Get the noise temperature (including antenna noise temperature)
rsFloat Receiver::GetNoiseTemperature(const SVec3 &angle) const
{
    return noise_temperature+Radar::GetNoiseTemperature(angle);
}

// Get the receiver noise temperature
rsFloat Receiver::GetNoiseTemperature() const
{
    return noise_temperature;
}

// Set the noise temperature of the receiver
void Receiver::SetNoiseTemperature(rsFloat temp)
{
    if (temp < -std::numeric_limits<rsFloat>::epsilon())
        throw std::runtime_error("Noise temperature set to negative value.");
    noise_temperature = temp;
}

// Set the length of the receive window
void Receiver::SetWindowProperties(rsFloat length, rsFloat prf, rsFloat skip)
{
    rsFloat rate = rsParameters::rate()*rsParameters::oversample_ratio();
    window_length = length;
    window_prf = prf;
    window_skip = skip;

    // The PRF and skip must be rounded to an even number of samples
    window_prf = 1/(std::floor(rate/window_prf)/rate);
    window_skip = std::floor(rate*window_skip)/rate;
}

// Return the number of responses
int Receiver::CountResponses() const
{
    return responses.size();
}

// Get the number of receive windows in the simulation time
int Receiver::GetWindowCount() const
{
    rsFloat time = rsParameters::end_time() - rsParameters::start_time();
    rsFloat pulses = time*window_prf;
    return static_cast<int>(std::ceil(pulses));
}

// Get the start time of the next window
rsFloat Receiver::GetWindowStart(int window) const
{
    // Calculate start time of pulse
    rsFloat stime = static_cast<rsFloat>(window)/window_prf+window_skip;

    // If there is timing jitter, add it
    if (!timing)
        throw std::logic_error("BUG: Receiver must be associated with timing source");
    return stime;
}

// Get the length of the receive window
rsFloat Receiver::GetWindowLength() const
{
    return window_length;
}

// Get the time skipped before the start of the receive window
rsFloat Receiver::GetWindowSkip() const
{
    return window_skip;
}

// Get the length of the receive window
rsFloat Receiver::GetPRF() const
{
    return window_prf;
}

// Set a flag
void Receiver::SetFlag(RecvFlag flag)
{
    flags |= flag;
}

// Check if a flag is set
bool Receiver::CheckFlag(RecvFlag flag) const
{
    return flags & flag;
}

///  Multipath dual functions

// Create a multipath dual of the given receiver
Receiver* rs::CreateMultipathDual(Receiver *recv, const MultipathSurface *surf)
{
    // If we already have a dual, simply return the pointer to it
    if (recv->dual)
        return recv->dual;

    // Get the dual platform
    Platform *dual_plat = CreateMultipathDual(recv->GetPlatform(), surf);

    // Create a new receiver object
    Receiver *dual = new Receiver(dual_plat, recv->GetName()+"_dual");

    // Assign the new receiver object to the current object
    recv->dual = dual;

    // Copy data from the Radar object
    dual->antenna = recv->antenna;
    if (recv->attached)
        dual->attached = CreateMultipathDual(dynamic_cast<Transmitter*>(const_cast<Radar*>(recv->attached)), surf);
    dual->SetMultipathDual(surf->GetFactor());

    // Copy data from the receiver object
    dual->noise_temperature = recv->noise_temperature;
    dual->window_length = recv->window_length;
    dual->window_prf = recv->window_prf;
    dual->window_skip = recv->window_skip;
    dual->timing = recv->timing;

    //Done, return the created object
    return dual;
}

// Create a multipath dual of the given transmitter
Transmitter* rs::CreateMultipathDual(Transmitter *trans, const MultipathSurface *surf)
{
    // If we already have a dual, simply return a pointer to it
    if (trans->dual)
        return trans->dual;

    // Get the dual platform
    Platform* dual_plat = CreateMultipathDual(trans->GetPlatform(), surf);

    // Create a new transmitter object
    Transmitter *dual = new Transmitter(dual_plat, trans->GetName()+"_dual", trans->pulsed);

    // Assign the the transmitter object to the current object
    trans->dual = dual;

    // Copy data from the Radar object
    dual->antenna = trans->antenna;
    if (trans->attached)
        dual->attached = CreateMultipathDual(dynamic_cast<Receiver*>(const_cast<Radar*>(trans->attached)), surf);
    dual->SetMultipathDual(surf->GetFactor());

    // Copy data from the transmitter object
    dual->prf = trans->prf;
    dual->pulsed = trans->pulsed;
    dual->signal = trans->signal;
    dual->timing = trans->timing;

    // Done, return the created object
    return dual;
}
