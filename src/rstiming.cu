#include "hip/hip_runtime.h"
/// rstiming.cpp - Implementation of timing sources
/// Marc Brooker, 16 October 2006
/// Edited by Yaaseen Martin, 27 August 2019

#include "rstiming.cuh"
#include "rsnoise.cuh"
#include "rsdebug.cuh"
#include <algorithm>

using namespace rs;

/// Timing Implementation

// Default constructor
Timing::Timing(const std::string &name):
    name(name)
{
}

// Destructor
Timing::~Timing()
{
}

// Get the name of the timing source
std::string Timing::GetName() const
{
    return name;
}

/// PrototypeTiming Implementation

// Constructor
PrototypeTiming::PrototypeTiming(const std::string &name):
    name(name)
{
    freq_offset = 0;
    phase_offset = 0;
    random_phase = 0;
    random_freq = 0;
    frequency = 0;
    synconpulse = false;
}

// Add an alpha and a weight to the timing prototype
void PrototypeTiming::AddAlpha(rsFloat alpha, rsFloat weight)
{
    alphas.push_back(alpha);
    weights.push_back(weight);
}

// Get the alphas and weights from the prototype
void PrototypeTiming::GetAlphas(std::vector<rsFloat> &get_alphas, std::vector<rsFloat> &get_weights) const
{
    // Copy the alpha and weight vectors
    get_alphas = alphas;
    get_weights = weights;
}

// Set a constant frequency offset
void PrototypeTiming::AddFreqOffset(rsFloat offset)
{
    if (random_freq)
        rsDebug::printf(rsDebug::RS_IMPORTANT, "[Important] Random frequency offset and constant frequency offset are set for timing source %s. Only the random offset will be used.\n", GetName().c_str());
    freq_offset = offset;
}

// Set a constant phase offset
void PrototypeTiming::AddPhaseOffset(rsFloat offset)
{
    if (random_phase)
        rsDebug::printf(rsDebug::RS_IMPORTANT, "[Important] Random phase offset and constant phase offset are set for timing source %s. Only the random offset will be used.\n", GetName().c_str());
    phase_offset = offset;
}

// Set a random frequency offset
void PrototypeTiming::AddRandomFreqOffset(rsFloat stdev)
{
    if (freq_offset)
        rsDebug::printf(rsDebug::RS_IMPORTANT, "[Important] Random frequency offset and constant frequency offset are set for timing source %s. Only the random offset will be used\n", GetName().c_str());
    random_freq = stdev;
}

// Set a random phase offset
void PrototypeTiming::AddRandomPhaseOffset(rsFloat stdev)
{
    if (phase_offset)
        rsDebug::printf(rsDebug::RS_IMPORTANT, "[Important] Random phase offset and constant phase offset are set for timing source %s. Only the random offset will be used.\n", GetName().c_str());
    random_phase = stdev;
}

// Get the phase offset
rsFloat PrototypeTiming::GetPhaseOffset() const
{
    if (random_phase != 0)
        return rsNoise::WGNSample(random_phase);
    else
        return phase_offset;
}

// Get the phase offset
rsFloat PrototypeTiming::GetFreqOffset() const
{
    if (random_freq != 0)
        return rsNoise::WGNSample(random_freq);
    else
        return freq_offset;
}

// Get the frequency
rsFloat PrototypeTiming::GetFrequency() const
{
    return frequency;
}


// Get the name of the prototype
std::string PrototypeTiming::GetName() const
{
    return name;
}

// Set the base frequency of the clock model
void PrototypeTiming::SetFrequency(rsFloat freq) {
    frequency = freq;
}

// Set the sync on pulse flag -- timing error resets at the start of the pulse
void PrototypeTiming::SetSyncOnPulse()
{
    synconpulse = true;
}

// Get the value of the sync on pulse flag
bool PrototypeTiming::GetSyncOnPulse() const
{
    return synconpulse;
}

/// ClockModelTiming Implementation

// Constructor
ClockModelTiming::ClockModelTiming(const std::string &name):
    Timing(name),
    enabled(false),
    model(0)
{
}

// Destructor
ClockModelTiming::~ClockModelTiming() {
    delete model;
}

// Initialize the clock model generator
void ClockModelTiming::InitializeModel(const PrototypeTiming *timing)
{
    if (!alphas.empty())
        throw std::logic_error("[BUG] ClockModelTiming::InitializeModel called more than once");

    // Copy the alpha and weight vectors
    timing->GetAlphas(alphas, weights);

    // Create the generator
    model = new ClockModelGenerator(alphas, weights, timing->GetFrequency(), timing->GetPhaseOffset(), timing->GetFreqOffset(), 15);

    // Warn if frequency is not set
    if (timing->GetFrequency() == 0.0)
        rsDebug::printf(rsDebug::RS_IMPORTANT, "WARNING: Timing source frequency not set, results could be incorrect.");

    // Get the carrier frequency
    frequency = timing->GetFrequency();

    // Get the sync on pulse flag
    synconpulse = timing->GetSyncOnPulse();

    // Enable the model
    enabled = true;

}

// Return the enabled state of the clock model
bool ClockModelTiming::Enabled()
{
    return enabled && model->Enabled();
}

// Get the real time of a particular pulse
rsFloat ClockModelTiming::GetPulseTimeError() const
{
    if (enabled)
        return model->GetSample();
    else
        return 0;
}

// Skip a sample, computing only enough to preserve long term correlations
void ClockModelTiming::SkipSamples(long long samples)
{
    if (enabled)
        model->SkipSamples(samples);
}

// Get the value of the sync on pulse flag
bool ClockModelTiming::GetSyncOnPulse() const
{
    return synconpulse;
}

// Reset the clock phase error to zero
void ClockModelTiming::Reset()
{
    model->Reset();
}

// Get the next sample of time error for a particular pulse
rsFloat ClockModelTiming::NextNoiseSample()
{
    if (enabled)
        return model->GetSample();
    else
        return 0;
}

// Get the carrier frequency of the modelled clock
rsFloat ClockModelTiming::GetFrequency() const
{
    return frequency;
}
