#include "hip/hip_runtime.h"
/// rspath.cpp - Implementation of rotation and position path classes
/// Marc Brooker, 21 April 2006
/// Edited by Yaaseen Martin, 02 September 2019

#include <cmath> // For fmod
#include <stdio.h>
#include <algorithm>
#include "rspath.cuh"
#include "rsdebug.cuh"
#include "rspython.cuh"
#include "rsmultipath.cuh"

using namespace rs;

// The interpolation functions are implemented as template functions to ease adding more functions to both rotation and motion classes

// Static "interpolation" function - the path is at the same point all the time
template <typename T> void GetPositionStatic(rsFloat t, T &coord, const std::vector<T> &coords) {
    if (coords.empty())
        throw PathException("coord list empty during GetPositionStatic");
    coord = coords[0];
}

// Linear interpolation function
template <typename T> void GetPositionLinear(rsFloat t, T &coord, const std::vector<T> &coords )
{
    T sKey;
    sKey = 0;
    sKey.t = t;
    typename std::vector<T>::const_iterator xrp;
    xrp = upper_bound(coords.begin(), coords.end(), sKey);

    // Check if we are over one of the end points
    if (xrp == coords.begin())
        coord = *xrp; // We are at the left endpoint
    else if (xrp == coords.end())
        coord = *(xrp-1); // We are at the right endpoint
    else { // We are at neither endpoint - perform linear interpolation
        int xri = xrp - coords.begin();
        int xli = xri-1;

        rsFloat iw = coords[xri].t - coords[xli].t;
        rsFloat rw = (coords[xri].t - t)/iw;
        rsFloat lw = 1-rw;

        // Insert the interpolated values in coord
        coord = coords[xri]*lw + coords[xli]*rw;
    }
  // Set the time part of the coordinate
  coord.t = t;
}

// Cubic spline interpolation function; method from "Numerical Recipes in C, Second Edition" by Press, et al. pages 114-116
template <typename T> void GetPositionCubic(rsFloat t, T &coord, const std::vector<T> &coords, const std::vector <T> &dd)
{
    T sKey;
    sKey = 0;
    sKey.t = t;
    typename std::vector<T>::const_iterator xrp;

    // Check that we are finalized, if not, complain
    xrp=upper_bound(coords.begin(), coords.end(), sKey);

    // Check if we are over one of the end points
    if (xrp == coords.begin())
        coord = *xrp; // We are at the left endpoint
    else if (xrp == coords.end())
        coord = *(xrp-1); // We are at the right endpoint
    else { // We are at neither endpoint - perform cubic spline interpolation
        int xri = xrp-coords.begin();
        int xli = xri - 1;
        rsFloat xrd = (coords[xri].t - t), xld = (t - coords[xli].t), iw = (coords[xri].t-coords[xli].t), iws=iw*iw/6.0;
        rsFloat A = xrd/iw, B=xld/iw, C=(A*A*A-A)*iws, D=(B*B*B-B)*iws;
        coord = coords[xli]*A + coords[xri]*B + dd[xli]*C + dd[xri]*D;
    }
    // Set the time part of the coordinate
    coord.t = t;
}

// Finalize function to calculate vector of second derivatives; method from "Numerical Recipes in C, Second Edition" by Press, et al. pages 114-116
template <typename T> void finalizeCubic(std::vector <T> &coords, std::vector <T> &dd) {
    int size = coords.size();
    std::vector <T> tmp(size);
    dd.resize(size);

    // Set the second derivative at the end points to zero
    dd[0] = 0;
    dd[size-1] = 0;
    // Forward pass of calculating the second derivatives at each point
    for (int i = 1; i < size-1; i++)
    {
        T yrd = coords[i+1]-coords[i], yld=coords[i]-coords[i-1];
        rsFloat xrd = coords[i+1].t-coords[i].t, xld=coords[i].t-coords[i-1].t;
        T dr = yrd/xrd;
        T dl = yld/xld;
        rsFloat iw = coords[i+1].t-coords[i-1].t;
        rsFloat si = xld/iw;
        T p = dd[i-1]*si+2.0;
        dd[i] = (si-1.0)/p;
        tmp[i] = ((yrd/xrd - yld/xld)*6.0/iw - tmp[i-1]*si)/p;
    }

    // Second (backward) pass of calculation
    for (int i = size-2; i >= 0; i--)
        dd[i] = dd[i]*dd[i+1]+tmp[i];
}

/// Path Implementation

Path::Path(Path::InterpType type):
    final(false), type(type)
{
    pythonpath = 0; // No Python path until loaded
}

void Path::AddCoord(Coord& coord) {
    std::vector < Coord > :: iterator iter;

    // Find the position to insert the coordinate, preserving sort
    iter = lower_bound(coords.begin(), coords.end(), coord);

    // Insert the new coordinate
    coords.insert(iter, coord);

    // We are not finalized if we have inserted a coord
    final = false;
}

// Get the position of the path object at a specified time
Vec3 Path::GetPosition(rsFloat t) const {
    Coord coord;
    if (!final)
        throw PathException("Finalize not called before GetPosition");

    //Call the interpolation function relevent to the type
    switch (type) {
    case RS_INTERP_STATIC:
        GetPositionStatic<Coord>(t, coord, coords);
        break;
    case RS_INTERP_LINEAR:
        GetPositionLinear<Coord>(t, coord, coords);
        break;
    case RS_INTERP_CUBIC:
        GetPositionCubic<Coord>(t, coord, coords, dd);
        break;
    case RS_INTERP_PYTHON:
        if (!pythonpath)
            throw std::logic_error("Python path GetPosition called before module loaded");
        return pythonpath->GetPosition(t);
    }

    //Return the position part of the result
    return coord.pos;
}

// Finalize the path - doing some once-per-path calculations if necessary
void Path::Finalize()
{
    if (!final) {
        switch (type) {
        case RS_INTERP_STATIC:
            break;
        case RS_INTERP_LINEAR:
            break;
        case RS_INTERP_CUBIC:
            finalizeCubic<Coord>(coords, dd);
            break;
        case RS_INTERP_PYTHON:
            break;
        }
        final = true;
    }
}

// Set the interpolation type of the path
void Path::SetInterp(InterpType settype)
{
    final = false;
    type = settype;
}

// Compares two paths at the same time and returns a vector with the distance and angle
SVec3 Compare(const rsFloat time, Path &start, Path &end)
{
    Vec3 difference = end.GetPosition(time)-start.GetPosition(time);
    SVec3 result(difference); // Get the result in spherical co-ordinates
    return result;
}

/// Load a python path function
void Path::LoadPythonPath(const std::string& modname, const std::string& pathname)
{
    // If we have one already, delete it
    if (pythonpath)
        delete pythonpath;

    // Load the new python path
    pythonpath = new rsPython::PythonPath(modname, pathname);
}

/// Create a new path which is a reflection of this one around the given plane
Path* rs::ReflectPath(const Path *path, const MultipathSurface *surf)
{
    // Do not support multipath on Python paths for now
    if (path->pythonpath)
        throw std::runtime_error("ERROR: Multipath surfaces are not currently supported for Python paths");

    // Create a new path object
    Path* dual = new Path(path->type);

    // Add all the coords from the current path to the old path, reflecting about the multipath plane
    std::vector<Coord>::const_iterator iter = path->coords.begin();
    for (; iter != path->coords.end(); iter++) {
        Coord refl;
        refl.t = (*iter).t;

        // Reflect the point in the plane
        refl.pos = surf->ReflectPoint((*iter).pos);
        rsDebug::printf(rsDebug::RS_VERBOSE, "Reflected (%g, %g, %g) to (%g, %g, %g)\n", (*iter).pos.x, (*iter).pos.y, (*iter).pos.z, refl.pos.x, refl.pos.y, refl.pos.z);
        dual->AddCoord(refl);
    }

    // Finalize the new path
    dual->Finalize();

    // Done, return the new path
    return dual;
}

/// RotationPath Implementation

RotationPath::RotationPath(RotationPath::InterpType type):
    final(false), start(0), rate(0), type(type)
{
}

void RotationPath::AddCoord(RotationCoord& coord) {
    std::vector < RotationCoord > :: iterator iter;

    // Find the position to insert the coordinate, preserving sort
    iter = lower_bound(coords.begin(), coords.end(), coord);

    // Insert the new coordinate
    coords.insert(iter, coord);

    // We are not finalized if we have inserted a coord
    final = false;
}

// Get the position of the path object at a specified time
SVec3 RotationPath::GetPosition(rsFloat t) const {
    RotationCoord coord;
    if (!final)
        throw PathException("Finalize not called before GetPosition in Rotation");

    // Call the interpolation function relevent to the type
    switch (type){
    case RS_INTERP_STATIC:
        GetPositionStatic<RotationCoord>(t, coord, coords);
        break;
    case RS_INTERP_LINEAR:
        GetPositionLinear<RotationCoord>(t, coord, coords);
        break;
    case RS_INTERP_CUBIC:
        GetPositionCubic<RotationCoord>(t, coord, coords, dd);
        break;
    case RS_INTERP_CONSTANT:
        coord.t = t;
        coord.azimuth = std::fmod(t*rate.azimuth+start.azimuth, static_cast<rsFloat>(2*M_PI));
        coord.elevation = std::fmod(t*rate.elevation+start.elevation, static_cast<rsFloat>(2*M_PI));
        break;
    }
    return SVec3(1, coord.azimuth, coord.elevation);
}

// Finalize the path - doing some once-per-path calculations if necessary
void RotationPath::Finalize()
{
    if (!final) {
        switch (type) {
        case RS_INTERP_STATIC:
            break;
        case RS_INTERP_LINEAR:
            break;
        case RS_INTERP_CONSTANT:
            break;
        case RS_INTERP_CUBIC:
            finalizeCubic<RotationCoord>(coords, dd);
            break;
        }
        final = true;
    }
}

// Set the interpolation type
void RotationPath::SetInterp(InterpType setinterp)
{
    type = setinterp;
    final = false;
}

// Set properties for fixed rate motion
void RotationPath::SetConstantRate(RotationCoord &setstart, RotationCoord &setrate)
{
    start = setstart;
    rate = setrate;
    type = RS_INTERP_CONSTANT;
    final = true;
}

/// Coord Implementation

// Component-wise multiplication of space coordinates
Coord rs::operator* (Coord a, Coord b)
{
    Coord c;
    c.pos = a.pos * b.pos;
    c.t = a.t; // Only multiply space coordinates
    return c;
}

// Component-wise addition of space coordinates
Coord rs::operator+ (Coord a, Coord b)
{
    Coord c;
    c.pos = a.pos;
    c.pos += b.pos;
    c.t = a.t; // Only add space coordinates
    return c;
}

// Component-wise subtraction of space coordinates
Coord rs::operator- (Coord a, Coord b)
{
    Coord c;
    c.pos = a.pos;
    c.pos -= b.pos;
    c.t = a.t;
    return c;
}

// Component-wise division of space coordinates
Coord rs::operator/ (const Coord &a, const Coord &b)
{
    Coord c;
    c.pos = a.pos / b.pos;
    c.t = a.t; // Only add space coordinates
    return c;
}

// Add a constant to a PathCoord
Coord rs::operator+ (Coord a, rsFloat b)
{
    Coord c;
    c.pos += b;
    c.t = a.t;
    return c;
}

// Multiply by a rsFloat constant
Coord rs::operator* (Coord a, rsFloat b)
{
    Coord c;
    c.pos = a.pos * b;
    c.t = a.t;
    return c;
}

Coord rs::operator/ (rsFloat a, Coord b)
{
    Coord c;
    c.pos = a / b.pos;
    c.t = b.t;
    return c;
}

Coord rs::operator/ (const Coord &b, rsFloat a)
{
    Coord c;
    c.pos = b.pos / a;
    c.t = b.t;
    return c;
}

/// RotationCoord Implementation

// Component-wise multiplication of space coordinates
RotationCoord rs::operator* (RotationCoord a, RotationCoord b)
{
    RotationCoord c;
    c.azimuth = a.azimuth*b.azimuth;
    c.elevation = a.elevation*b.elevation;
    c.t = a.t; // Only multiply space coordinates
    return c;
}

// Component-wise addition of space coordinates
RotationCoord rs::operator+ (RotationCoord a, RotationCoord b)
{
    RotationCoord c;
    c.azimuth = a.azimuth+b.azimuth;
    c.elevation = a.elevation+b.elevation;
    c.t = a.t; // Only add space coordinates
    return c;
}

// Component-wise subtraction of space coordinates
RotationCoord rs::operator- (RotationCoord a, RotationCoord b)
{
    RotationCoord c;
    c.azimuth = a.azimuth-b.azimuth;
    c.elevation = a.elevation-b.elevation;
    c.t = a.t;
    return c;
}

// Component-wise division of space coordinates
RotationCoord rs::operator/ (RotationCoord a, RotationCoord b)
{
    RotationCoord c;
    c.azimuth = a.azimuth/b.azimuth;
    c.elevation = a.elevation/b.elevation;
    c.t = a.t; // Only add space coordinates
    return c;
}

// Add a constant to a PathRotationCoord
RotationCoord rs::operator+ (RotationCoord a, rsFloat b)
{
    RotationCoord c;
    c.azimuth = a.azimuth+b;
    c.elevation = a.elevation+b;
    c.t = a.t;
    return c;
}

// Multiply by a rsFloat constant
RotationCoord rs::operator* (RotationCoord a, rsFloat b)
{
    RotationCoord c;
    c.azimuth = a.azimuth*b;
    c.elevation = a.elevation*b;
    c.t = a.t;
    return c;
}

RotationCoord rs::operator/ (rsFloat a, RotationCoord b)
{
    RotationCoord c;
    c.azimuth = a/b.azimuth;
    c.elevation = a/b.elevation;
    c.t = b.t;
    return c;
}

RotationCoord rs::operator/ (RotationCoord b, rsFloat a)
{
    RotationCoord c;
    c.azimuth = b.azimuth/a;
    c.elevation = b.elevation/a;
    c.t = b.t;
    return c;
}

/// Create a new path which is a reflection of this one around the given plane
RotationPath* rs::ReflectPath(const RotationPath *path, const MultipathSurface *surf)
{
    // Create the new RotationPath object
    RotationPath *dual = new RotationPath(path->type);

    // Copy constant rotation params
    dual->start = path->start;
    dual->rate = path->rate;

    // Copy the coords, reflecting them in the surface
    std::vector<RotationCoord>::const_iterator iter = path->coords.begin();
    for (; iter != path->coords.end(); iter++)
    {
        RotationCoord rc;

        // Time copies directly
        rc.t = (*iter).t;
        SVec3 sv(1, (*iter).azimuth, (*iter).elevation);
        Vec3 v(sv);

        //Reflect the point in the given plane
        v = surf->ReflectPoint(v);
        SVec3 refl(v);
        rc.azimuth = refl.azimuth;
        rc.elevation = refl.elevation;
        dual->AddCoord(rc);
    }

    //Finalize the copied path
    dual->Finalize();

    //Done, return the created object
    return dual;
}
