#include "hip/hip_runtime.h"
/// rsnoise.cpp - Functions for generating different types of noise
/// Marc Brooker, 14 August 2006
/// Edited by Yaaseen Martin, 27 August 2019

#include <cmath>
#include <limits>
#include <boost/random.hpp>
#include "rsnoise.cuh"
#include "rsdebug.cuh"
#include "rsparameters.cuh"
#include "rsdsp.cuh"

using namespace rs;

namespace {
    // Use the Mersenne Twister PRNG with parameter "19937"
    // Mersenne Twister: A 623-dimensionally equidistributed uniform pseudo-random number generator
    // Matsumoto et al., ACM Transactions on Modeling and Computer Simulation, January 1998
    boost::mt19937* rng;

    //Used to generate single noise samples
    boost::normal_distribution<rsFloat> nd(0,1);
    boost::uniform_real<rsFloat> ud(0, 1.0);
    boost::variate_generator<boost::mt19937&, boost::normal_distribution<rsFloat> >* normal_vg;
    boost::variate_generator<boost::mt19937&, boost::uniform_real<rsFloat> >* uniform_vg;
}

/// Implementation of non-class functions

// Initialize the random number generator code (must be called once, after the loading of the script)
void rsNoise::InitializeNoise()
{
    delete rng;
    delete normal_vg;
    rng = new boost::mt19937(rsParameters::random_seed());
    normal_vg = new boost::variate_generator<boost::mt19937&, boost::normal_distribution<rsFloat> >(*rng, nd);
    uniform_vg = new boost::variate_generator<boost::mt19937&, boost::uniform_real<rsFloat> >(*rng, ud);
}

// Clean up the noise code
void rsNoise::CleanUpNoise()
{
    delete rng;
    delete normal_vg;
    delete uniform_vg;
}

// Return a single sample of white gaussian noise
rsFloat rsNoise::WGNSample(rsFloat stddev)
{
    if (stddev > std::numeric_limits<rsFloat>::epsilon())
        return (*normal_vg)()*stddev;
    else
        return 0;
}

// Return a single uniformly distributed sample in [0, 1]
rsFloat rsNoise::UniformSample()
{
    return (*uniform_vg)();
}

// Calculate noise amplitude from the temperature
rsFloat rsNoise::NoiseTemperatureToPower(rsFloat temperature, rsFloat bandwidth)
{
    return rsParameters::boltzmann_k()*temperature*bandwidth; //See equations.tex
}

/// NoiseGenerator Implementation

//Constructor
NoiseGenerator::NoiseGenerator()
{
}

//Destructor
NoiseGenerator::~NoiseGenerator()
{
}

/// Gamma Generator

// Constructor
GammaGenerator::GammaGenerator(rsFloat k):
    dist(k),
    gen(*rng, dist)
{
}

// Destructor
GammaGenerator::~GammaGenerator()
{
}

// Get a single random sample
rsFloat GammaGenerator::GetSample()
{
    return gen();
}

// Operator to get a random sample
rsFloat GammaGenerator::operator()()
{
    return gen();
}

/// WGNGenerator Implementation

// Constructor
WGNGenerator::WGNGenerator(rsFloat stddev)
{
    dist = boost::normal_distribution<rsFloat>(0, stddev);
    gen = new boost::variate_generator<boost::mt19937&, boost::normal_distribution<rsFloat> >(*rng, dist);
}

// Default constructor
WGNGenerator::WGNGenerator()
{
    dist = boost::normal_distribution<rsFloat>(0, 1);
    gen = new boost::variate_generator<boost::mt19937&, boost::normal_distribution<rsFloat> >(*rng, dist);
}

// Destructor
WGNGenerator::~WGNGenerator()
{
    delete gen;
}

// Get a sample from the rng
rsFloat WGNGenerator::GetSample()
{
    return (*gen)();
}

/// FAlphaBranch Implementation

// Constructor
FAlphaBranch::FAlphaBranch(rsFloat ffrac, unsigned int fint, FAlphaBranch *pre, bool last):
    pre(pre),
    last(last),
    ffrac(ffrac),
    fint(fint)
{
    rsDebug::printf(rsDebug::RS_VERY_VERBOSE, "Making branch ffrac=%f fint=%d\n", ffrac, fint);

    //Calculate scale for upsampling
    upsample_scale = std::pow(10, ffrac+fint+0.5);

    //Initialize the filters for shaping, highpass and upsampling
    Init();

    // Create a buffer for ten samples
    buffer = new rsFloat[10];
    if (!last)
        Refill();
}

// Destructor
FAlphaBranch::~FAlphaBranch() {
    delete pre;
    Clean();
}

// Initialize the branch filters
void FAlphaBranch::Init() {
    shape_filter = 0;
    integ_filter = 0;
    highpass = 0;

    //Create the upsampler
    upsampler = new DecadeUpsampler();
    if (pre){
        // Numerator coefficients for elliptical highpass
        const rsFloat hp_num[12] =
        {
            3.817871081981451e-01,
            -4.093384095523618e+00,
            2.005300512623078e+01,
            -5.924672881811163e+01,
            1.172948159891025e+02,
            -1.633810410083022e+02,
            1.633810410083034e+02,
            -1.172948159891052e+02,
            5.924672881811390e+01,
            -2.005300512623186e+01,
            4.093384095523903e+00,
            -3.817871081981776e-01
        };

        // Denominator coefficients for elliptical highpass
        const rsFloat hp_den[12] =
        {
            1.000000000000000e+00,
            -8.829695665523831e+00,
            3.583068809011030e+01,
            -8.811479652970442e+01,
            1.457874067329429e+02,
            -1.702715637111961e+02,
            1.431504350055831e+02,
            -8.656925883534657e+01,
            3.687395592491803e+01,
            -1.052413841411803e+01,
            1.808292123637038e+00,
            -1.412932578340511e-01
        };

        //Initialize the highpass filter
        highpass = new IIRFilter(hp_den, hp_num, 12);
    }

    // Initialize the shaping filter
    if (ffrac == 0.5) {
        // Numerator co-efficients for 1/f^0.5 rolloff
        const rsFloat sf_num[16] =
        {
            5.210373977738306e-03,
            -7.694671394585578e-03,
            1.635979377907092e-03,
            9.852449140857658e-05,
            -2.080553126780113e-03,
            4.088764157029523e-03,
            -1.549082440084623e-03,
            9.054734252370680e-04,
            -3.467369912368729e-04,
            4.516383087838856e-04,
            -1.063356106118517e-03,
            1.330008998057684e-04,
            6.556909567323943e-04,
            -4.839476350293955e-04,
            6.664936170526832e-05,
            1.528520559763056e-05
        };

        const rsFloat sf_den[16] =
        {
            1.000000000000000e+00,
            -2.065565041154101e+00,
            1.130909190864681e+00,
            -1.671244644503288e-01,
            -3.331474931013877e-01,
            9.952625337612708e-01,
            -7.123036343635182e-01,
            3.297062696290504e-01,
            -1.925691520710595e-01,
            1.301247006176314e-01,
            -2.702016290409912e-01,
            1.455380885858886e-01,
            1.091921868353888e-01,
            -1.524953111510459e-01,
            5.667716332023935e-02,
            -2.890314873767405e-03
        };

        // Gain of shaping filter
        shape_gain = 5.210373977738306e-03;

        // Create the shaping filter
        shape_filter = new IIRFilter(sf_den, sf_num, 16);
    }
    else if (ffrac == 0) {
        shape_filter = 0;
    }
    else {
        rsDebug::printf(rsDebug::RS_CRITICAL, "Value of ffrac is %f\n", ffrac);
        throw std::runtime_error("Fractional integrator values other than 0.5 not currently supported");
    }

    // Initialize the integration filters
    if (fint > 0) {
        // Gain of integration filter
        integ_gain = 1;
        if (fint == 1) {
            const rsFloat i_den[2] = {1, -1};
            const rsFloat i_num[2] = {1, 0};
            integ_filter = new IIRFilter(i_den, i_num, 2);
        }
        if (fint == 2) {
            const rsFloat i_den[3] = {1, -2, 1};
            const rsFloat i_num[3] = {1, 0, 0};
            integ_filter = new IIRFilter(i_den, i_num, 3);
        }
        if (fint > 2)
          throw std::runtime_error("Only alpha values between 2 and -2 are supported for noise generation");
    }

    // Initialize the offset
    offset_sample = 0;
    got_offset = false;

    // Create a buffer for ten samples
    buffer = new rsFloat[10];
    if (!last)
        Refill();
    pre_scale = 1;
}

// Get a sample from the branch
rsFloat FAlphaBranch::GetSample() {
    if (!last) {
        rsFloat ret = buffer[buffer_samples];
        buffer_samples++;
        if (buffer_samples == 10)
            Refill();
        return ret;
    }
    else {
        return CalcSample()+offset_sample*upsample_scale;
    }
}

// Clean up the filters, etc
void FAlphaBranch::Clean() {
    delete highpass;
    delete[] buffer;
    delete integ_filter;
    delete shape_filter;
    delete upsampler;
}

// Calculate a single sample
rsFloat FAlphaBranch::CalcSample() {
    rsFloat sample = rsNoise::WGNSample(1);
    if (shape_filter)
        sample = shape_filter->Filter(sample)/shape_gain;
    if (integ_filter)
        sample = integ_filter->Filter(sample)/integ_gain;
    if (pre) {
        // Apply highpass only if we have branches below us
        sample = highpass->Filter(sample);

        // If there is a branch below us, add a sample from that
        if (got_offset) {
            sample += pre->GetSample()*pre_scale-offset_sample;
        }
        else {
            got_offset = true;
            offset_sample = pre->GetSample()*pre_scale;
        }
    }
    return sample;
}

// Refill the buffer
void FAlphaBranch::Refill() {
    rsFloat sample = CalcSample();

    // Pass the sample to the upsampler
    upsampler->Upsample(sample, buffer);

    // Scale the buffer
    for (int i = 0; i < 10; i++) {
        buffer[i] *= upsample_scale;
        buffer[i] += offset_sample;
    }

    // Reset the sample counter
    buffer_samples = 0;
}

// Refill the buffer
void FAlphaBranch::Flush(rsFloat scale = 1.0) {
    Clean();
    Init();
    pre_scale = scale;
}

/// MultirateGenerator Implementation

// Constructor
MultirateGenerator::MultirateGenerator(rsFloat alpha, unsigned int branches)
{
    rsFloat beta = -(alpha - 2)/2.0;

    //Calculate the integer and fractional parts of beta
    int fint = (int)std::floor(beta);
    rsFloat ffrac = fmod(beta, 1);

    //Build the multirate filter tree
    CreateTree(ffrac, fint, branches);
    scale = 1.0/std::pow(10.0, (-alpha+2)*2.0);
}

// Destructor
MultirateGenerator::~MultirateGenerator()
{
    delete topbranch;
}

// Get a single noise sample
rsFloat MultirateGenerator::GetSample()
{
    return topbranch->GetSample()*scale;
}

// Skip a number of samples, preseverving correlations
void MultirateGenerator::SkipSamples(long long samples)
{
    std::vector<FAlphaBranch*> flushbranches;
    int skip_branches = (int)std::floor(std::log10(samples))-1;
    if (skip_branches > 0) {
        FAlphaBranch *branch = topbranch;
        for (int i = 0; (i < skip_branches) && (branch != 0); i++) {
            flushbranches.push_back(branch);
            branch = branch->pre;
        }
        if (branch) {
            // Now generate the samples of the lower branches
            samples = (int)(samples/std::pow(10.0, (double)skip_branches));

            for (int i = 0; i < samples; i++)
            branch->GetSample();
        }

        // Flush the buffers of the upper branches
        int size = flushbranches.size();
        flushbranches[size-1]->Flush(std::pow(10.0, skip_branches-2.0));
        for (int i = size-2; i >= 0; i--)
            flushbranches[i]->Flush();
    }
    else {
        for (int i = 0; i < samples; i++)
            topbranch->GetSample();
    }
}

/// Create the branches of the filter structure tree stored as a linked list, with each link being a branch
void MultirateGenerator::CreateTree(rsFloat ffrac, int fint, unsigned int branches)
{
    if (branches == 0)
        throw std::runtime_error("Cannot create multirate noise generator with zero branches");

    // If ffrac and fint are both zero, we only need a single branch
    if ((ffrac == 0) && (fint == 0)) {
        topbranch = new FAlphaBranch(0, 0, 0, true);
    }
    else {
        topbranch = 0;
        for (unsigned int i = 0; i < branches-1; i++)
            topbranch = new FAlphaBranch(ffrac, fint, topbranch, false);
        topbranch = new FAlphaBranch(ffrac, fint, topbranch, true);
    }
}

/// Reset the output to zero
void MultirateGenerator::Reset()
{
    std::vector<FAlphaBranch*> flushbranches;

    //Build a reverse order list of branches
    FAlphaBranch *branch = topbranch;
    while (branch) {
        flushbranches.push_back(branch);
        branch = branch->pre;
    }

    // Flush the branch buffers in reverse order
    int size = flushbranches.size();
    for (int i = size-1; i >= 0; i--)
        flushbranches[i]->Flush();
}

/// ClockModelGenerator Implementation

// Constructor
ClockModelGenerator::ClockModelGenerator(const std::vector<rsFloat> &alpha, const std::vector<rsFloat> &in_weights, rsFloat frequency, rsFloat phase_offset, rsFloat freq_offset, int branches):
    phase_offset(phase_offset),
    freq_offset(freq_offset),
    frequency(frequency)
{
    weights = in_weights;
    std::vector<rsFloat>::const_iterator iter = alpha.begin();
    std::vector<rsFloat>::iterator witer = weights.begin();

    // Create the generators for each band
    for (; iter != alpha.end(); iter++, witer++) {
        MultirateGenerator *mgen = new MultirateGenerator(*iter, branches);
        generators.push_back(mgen);

        //Calibrate the weights using the measured calibration numbers
        if (*iter == 2) {
            *witer *= std::pow(10.0, 1.2250);
        }
        else if (*iter == 1) {
            *witer *= std::pow(10.0, 0.25);
        }
        else if (*iter == 0) {
            *witer *= std::pow(10.0, -0.25);
        }
        else if (*iter == -1) {
            *witer *= std::pow(10.0, -0.5);
        }
        else if (*iter == -2) {
            *witer *= std::pow(10.0, -1);
        }
    }
  count = 0;
}

// Destructor
ClockModelGenerator::~ClockModelGenerator()
{
    std::vector<MultirateGenerator *>::iterator iter;
    for (iter = generators.begin(); iter != generators.end(); iter++)
        delete *iter;
}

// Get a single noise sample
rsFloat ClockModelGenerator::GetSample()
{
    rsFloat sample = 0;

    // Get noise from the multirate generators for each band
    int size = generators.size();
    for (int i = 0; i < size; i++) {
        sample += generators[i]->GetSample()*weights[i];
    }

    // Add the phase and frequency offsets
    sample += phase_offset;

    // Calculate the count in clock frequencies
    sample += 2*M_PI*freq_offset*count/rsParameters::rate();
    count++;
    return sample;
}

// Skip some noise samples, calculating only the branches required to preserve correlations
void ClockModelGenerator::SkipSamples(long long samples)
{
    int gens = generators.size();
    for (int i = 0; i < gens; i++)
        generators[i]->SkipSamples(samples);
    count += samples;
}

// Reset the noise to zero
void ClockModelGenerator::Reset()
{
    int gens = generators.size();
    for (int i = 0; i < gens; i++)
        generators[i]->Reset();
    count = 0;
}

// Is the generator going to produce non-zero samples?
bool ClockModelGenerator::Enabled() {
    if ((!generators.empty()) || (freq_offset != 0) || (phase_offset != 0))
        return true;
    else
        return false;
}

/// PythonNoiseGenerator Implementation

// Constructor
PythonNoiseGenerator::PythonNoiseGenerator(const std::string& module, const std::string& function):
    generator(module, function)
{
}

// Destructor
PythonNoiseGenerator::~PythonNoiseGenerator()
{
}

// Get a single noise sample
rsFloat PythonNoiseGenerator::GetSample()
{
    return generator.GetSample();
}
