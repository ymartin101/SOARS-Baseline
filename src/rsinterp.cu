#include "hip/hip_runtime.h"
/// rsinterp.cpp - Implements interpolation class
/// Marc Brooker, 11 June 2007
/// Edited by Yaaseen Martin, 27 August 2019

#include <map>
#include <utility>
#include <stdexcept>
#include <cmath>
#include "rsinterp.cuh"
#include "rsdebug.cuh"

using std::map;
using std::vector;
using std::pair;

using namespace rs;

/// InterpSetData Implementation

// Data storage class for the InterpSet class
class rs::InterpSetData {
public:
    // Load samples into the set
    void LoadSamples(const vector<rsFloat> &x, const vector<rsFloat> &y);

    // Load a single sample into the set
    void InsertSample(rsFloat x, rsFloat y);

    // Get the interpolated value at a given point
    rsFloat Value(rsFloat x);

    // Get the maximum value in the set
    rsFloat Max() const;

    // Divide the set by a given number
    void Divide(rsFloat a);

private:
    std::map<rsFloat, rsFloat> data;
};


/// Load samples into the set
void InterpSetData::LoadSamples(const vector<rsFloat> &x, const vector<rsFloat> &y)
{
    vector<rsFloat>::const_iterator ix = x.begin();
    vector<rsFloat>::const_iterator iy = y.begin();
    for (; (ix != x.end()) && (iy != y.end()); ix++, iy++)
    {
        data.insert(pair<rsFloat, rsFloat>(*ix, *iy));
    }
}

/// Load a single sample into the set
void InterpSetData::InsertSample(rsFloat x, rsFloat y)
{
    data.insert(pair<rsFloat, rsFloat>(x, y));
}

/// Get the interpolated value for the given point
rsFloat InterpSetData::Value(rsFloat x)
{
    // Use linear interpolation, for now
    // If the map is empty, throw an exception and whine
    if (data.empty())
        throw std::logic_error("[BUG] Interpolation on an empty list in InterpSet");

    // Get the first element with a key greater than k
    map<rsFloat, rsFloat>::const_iterator iter = data.lower_bound(x);

    // If we are at the beginning of the set, return the value
    if (iter == data.begin())
        return (*iter).second;
    map<rsFloat, rsFloat>::const_iterator prev = iter;
    prev--;

    //If we are over the end, return the last value
    if (iter == data.end())
        return (*(prev)).second;

    //If we hit a sample exactly, return the value
    else if ((*iter).first == x)
        return (*iter).second;

    //Perform linear interpolation
    else {
        rsFloat x1 = (*prev).first;
        rsFloat x2 = (*iter).first;
        rsFloat y1 = (*prev).second;
        rsFloat y2 = (*iter).second;
        return y2*(x-x1)/(x2-x1)+y1*(x2-x)/(x2-x1);
    }
}

/// Get the maximum value in the set
rsFloat InterpSetData::Max() const
{
    map<rsFloat, rsFloat>::const_iterator iter;
    rsFloat max = 0;

    // Scan through the map, updating the maximum
    for (iter = data.begin(); iter != data.end(); iter++) {
        if (std::fabs((*iter).second) > max)
            max = std::fabs((*iter).second);
    }
    return max;
}

/// Divide the set by a given number
void InterpSetData::Divide(rsFloat a)
{
    map<rsFloat, rsFloat>::iterator iter;
    for (iter = data.begin(); iter != data.end(); iter++)
        (*iter).second /= a;
}

/// InterpSet Implementation

// Constructor
InterpSet::InterpSet()
{
    data = new InterpSetData();
}


// Destructor
InterpSet::~InterpSet()
{
    delete data;
}

/// Load a number of samples into the set
void InterpSet::LoadSamples(const std::vector<rsFloat> &x, const std::vector<rsFloat> &y)
{
    data->LoadSamples(x,y);
}

/// Load a single sample into the set
void InterpSet::InsertSample(rsFloat x, rsFloat y)
{
    data->InsertSample(x,y);
}

/// Get the interpolated value at the given point
rsFloat InterpSet::Value(rsFloat x)
{
    return data->Value(x);
}

/// Get the maximum value in the set
rsFloat InterpSet::Max() const
{
    return data->Max();
}

/// Divide every sample in the set by a given number
void InterpSet::Divide(rsFloat a) {
    data->Divide(a);
}
