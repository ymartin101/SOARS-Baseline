/// rspolarize.cpp - Implementation of polarization functions
/// Marc Brooker, 31 March 2008
/// Edited by Yaaseen Martin, 27 August 2019

#include "rspolarize.cuh"

using namespace rs;

/// JonesVector Implementation

// Constructor
JonesVector::JonesVector(std::complex<rsFloat> h, std::complex<rsFloat> v):
    h(h),
    v(v)
{
}

// Copy constructor
JonesVector::JonesVector(const JonesVector &iv):
    h(iv.h),
    v(iv.v)
{
}

// Assignment operator
JonesVector& JonesVector::operator= (const JonesVector &iv)
{
    v = iv.v;
    h = iv.h;
    return *this;
}

// Multiplication operator
JonesVector JonesVector::operator* (const PSMatrix &mat)
{
    JonesVector j(h*mat.s[0] + v*mat.s[1], h*mat.s[2] + v*mat.s[3]);
    return j;
}

/// PSMatrix Implementation

// Default constructor creates identity PSM
PSMatrix::PSMatrix()
{
    s[0] = s[3] = 1;
    s[1] = s[2] = 0;
}

  // Constructor
PSMatrix::PSMatrix(rsFloat s11, rsFloat s12, rsFloat s21, rsFloat s22)
{
    s[0] = s11;
    s[1] = s12;
    s[2] = s21;
    s[3] = s22;
}

// Copy constructor
PSMatrix::PSMatrix(const PSMatrix &im)
{
    for (int i = 0; i < 4; i++)
        s[i] = im.s[i];
}

// Assignment operator
PSMatrix& PSMatrix::operator= (const PSMatrix &im)
{
    for (int i = 0; i < 4; i++)
        s[i] = im.s[i];
    return *this;
}

// Dot product of two Jones vectors
std::complex<rsFloat> dot(const JonesVector &a, const JonesVector &b)
{
    return a.v*b.v+a.h*b.h;
}
