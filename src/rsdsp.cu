#include "hip/hip_runtime.h"
/// Digital Signal Processing support functions
/// Marc Brooker, 30 July 2007
/// Edited by Yaaseen Martin, 27 August 2019

#include "rsdsp.cuh"
using namespace rs;

#include <stdexcept>
#include <cmath>
#include <cstring>
#include <string.h>
#include "rsdebug.cuh"
#include "rsparameters.cuh"

/// Support Functions
namespace {
    // Calculate sin(pi * x) / (pi*x)
    rsFloat Sinc(rsFloat x) {
    if (x == 0)
        return 1.0;
    return std::sin(x*M_PI)/(x*M_PI);
    }

    /// Create a FIR filter using the Blackman window
    rsFloat *BlackmanFIR(rsFloat cutoff, int &length) {
        // Use double the render filter length, for faster rolloff than the render filter
        length = rsParameters::render_filter_length() * 2;
        rsFloat *coeffs = new rsFloat[length];
        rsFloat N = length / 2.0;
        for (int i = 0; i < length; i++) {
            rsFloat filt = Sinc(cutoff*(i - N));

            // Use the Blackman window for a suitable tradeoff between rolloff and stopband attenuation
            // Equivalent Kaiser beta = 7.04 (Oppenhiem and Schaffer, Hamming)
            rsFloat window = 0.42 - 0.5*cos(M_PI*i/N) + 0.08*cos(2*M_PI*i/N);
            coeffs[i] = filt*window;
        }
        return coeffs;
    }
}

/// Upsample size samples stored *in by an integer ratio and store the result in (pre-allocated) out
// TODO: this would be better as a multirate upsampler
// In fact, the whole scheme is currently sub-optimal - we could use better filters, better windows and a better approach
// it works okay for now, but users seeking higher accuracy can oversample outside SOARS until this is fixed
void rs::Upsample(const rsComplex *in, int size, rsComplex *out, int ratio)
{
    // Design the FIR filter for de-imaging
    int filt_length;
    rsFloat *coeffs = BlackmanFIR(1/rsFloat(ratio), filt_length);

    // Temporary buffer for zero padding and results
    rsComplex *tmp = new rsComplex[size*ratio+filt_length];
    for (int i = 0; i < size*ratio+filt_length; i++)
        tmp[i] = 0;

    // Stuff the data with a suitable number of zeros
    for (int i = 0; i < size; i++) {
        tmp[i*ratio] = in[i];
        for (int j = 1; j < (ratio-1); j++)
            tmp[i*ratio+j] = 0;
    }

    // Create a FIR filter object
    FIRFilter filt(coeffs, filt_length);
    filt.Filter(tmp, size*ratio+filt_length);

    // Copy results to output buffer
    for (int i = 0; i < size*ratio; i++) {
        out[i] = tmp[i+filt_length/2-1];
    }

    // Clean up
    delete[] tmp;
    delete[] coeffs;

}

/// Upsample size samples stored *in by an integer ratio and store the result in (pre-allocated) out
// TODO: This would be better (and much faster) as a multirate downsampler
void rs::Downsample(const rsComplex *in, int size, rsComplex *out, int ratio)
{
    // Design the FIR filter for anti-aliasing
    int filt_length;
    rsFloat *coeffs = BlackmanFIR(1/rsFloat(ratio), filt_length);

    // Temporary buffer for zero padding and results
    rsComplex *tmp = new rsComplex[size+filt_length];
    for (int i = size-1; i < size+filt_length; i++)
    tmp[i] = 0;

    // Copy the input into the temporary buffer, leaving zero padding at the end
    for (int i = 0; i < size; i++)
        tmp[i] = in[i];

    // Run the anti aliasing filter on the data
    FIRFilter filt(coeffs, filt_length);
    filt.Filter(tmp, size+filt_length);

    // Copy the results to the output buffer
    for (int i = 0; i < size/ratio; i++)
        out[i] = tmp[i*ratio+filt_length/2]/rsFloat(ratio);

    // Clean up
    delete[] coeffs;
    delete[] tmp;
}

/// Filter Implementation

// Constructor
DSPFilter::DSPFilter()
{
}

// Destructor
DSPFilter::~DSPFilter()
{
}

/// IIRFilter Implementation

// Constructor
IIRFilter::IIRFilter(const std::vector<rsFloat> &den_coeffs, const std::vector<rsFloat> &num_coeffs)
{
    // Get the filter order
    order = den_coeffs.size();

    // Check the filter order
    if (order != num_coeffs.size())
        throw std::logic_error("IIRFilter does not currently support mixed order filters");

    //Allocate memory to store co-efficients and state
    a = new rsFloat[order];
    b = new rsFloat[order];
    w = new rsFloat[order];

    //Load the co-efficients from the vectors into the arrays
    for (unsigned int i = 0; i < order; i++) {
        a[i] = den_coeffs[i];
        b[i] = num_coeffs[i];
        w[i] = 0;
    }
}

// Constructor
IIRFilter::IIRFilter(const rsFloat *den_coeffs, const rsFloat *num_coeffs, unsigned int order):
    order(order)
{
    a = new rsFloat[order];
    b = new rsFloat[order];
    w = new rsFloat[order];

    // Load the coefficients into the arrays
    for (unsigned int i = 0; i < order; i++) {
        a[i] = den_coeffs[i];
        b[i] = num_coeffs[i];
        w[i] = 0;
    }
}

// Destructor
IIRFilter::~IIRFilter()
{
    // Clean up the co-efficients and state
    delete[] a;
    delete[] b;
    delete[] w;
}

/// Pass a single sample through the filter
rsFloat IIRFilter::Filter(rsFloat sample)
{
    // Shift the filter state
    for (unsigned int j = order-1; j > 0; j--)
        w[j] = w[j-1];

    // Calculate w[0]
    w[0] = sample;
    for (unsigned int j = 1; j < order; j++)
        w[0] -= a[j]*w[j];

    // Calculate y[n]
    rsFloat out = 0;
    for (unsigned int j = 0; j < order; j++)
        out += b[j]*w[j];
    return out;
}

/// Pass an array of samples through the filter, filtering in place
void IIRFilter::Filter(rsFloat *samples, int size)
{
    for (int i = 0; i < size; i++)
    {
        // Shift the filter state
        for (unsigned int j = order-1; j > 0; j--)
            w[j] = w[j-1];

        // Calculate w[0]
        w[0] = samples[i];
        for (unsigned int j = 1; j < order; j++)
            w[0] -= a[j]*w[j];

        // Calculate y[n]
        samples[i] = 0;
        for (unsigned int j = 0; j < order; j++)
            samples[i] += b[j]*w[j];
    }
}

/// FIRFilter implementation

// Constructor
FIRFilter::FIRFilter(const std::vector<rsFloat> &coeffs)
{
    // Get the filter order
    order = coeffs.size();

    // Allocate memory to store co-efficients and state
    filter = new rsFloat[order];
    w = new rsFloat[order];

    // Load the co-efficients from the vectors into the arrays
    for (unsigned int i = 0; i < order; i++) {
        filter[i] = coeffs[i];
        w[i] = 0;
    }
}

// Constructor from coeffs
FIRFilter::FIRFilter(const rsFloat* coeffs, int count) {
    order = count;

    // Allocate memory to store co-efficients and state
    filter = new rsFloat[order];
    w = new rsFloat[order];

    // Load the co-efficients
    for (unsigned int i = 0; i < order; i++) {
        filter[i] = coeffs[i];
        w[i] = 0;
    }
}

// Destructor
FIRFilter::~FIRFilter()
{
    // Clean up memory
    delete[] filter;
    delete[] w;
}

/// Pass a single sample through the filter
inline rsFloat FIRFilter::Filter(rsFloat sample)
{
    return 0;
}

/// Pass an array of samples through the filter, filtering in place
// See Oppenheim and Scaffer, section 6.5 "Basic Network Structures for FIR Systems"
// TODO: Implement one of the more efficient FIR filter forms
inline void FIRFilter::Filter(rsFloat *samples, int size)
{
    // Allocate memory for a delay line, equal to the filter length
    rsFloat* line = new rsFloat[order];
    std::memset(line, 0, sizeof(rsFloat)*order);

    // Perform the inplace convolution with the pulse
    for (int i = 0; i < size; i++) {
        line[0] = samples[i];
        rsFloat res = 0;
        for (unsigned int j = 0; j < order; j++)
            res += line[order-j-1]*filter[j];
        samples[i] = res;

        // Move the line over by one sample
        for (int j = order; j > 0; j--)
            line[j] = line[j-1];
    }

    // Clean up
    delete[] line;
}

/// Pass an array of complex samples through the filter, filtering in place
inline void FIRFilter::Filter(std::complex<rsFloat> *samples, int size)
{
    // Allocate memory for a delay line, equal to the filter length
    rsComplex* line = new rsComplex[order];
    for (unsigned int i = 0; i < order; i++)
        line[i] = 0;

    // Perform the inplace convolution with the pulse
    for (int i = 0; i < size; i++) {
        line[0] = samples[i];
        rsComplex res = 0;
        for (unsigned int j = 0; j < order; j++)
            res += line[order-j-1]*filter[j];
        samples[i] = res;

        //Move the line over by one sample
        for (int j = order-1; j > 0; j--)
            line[j] = line[j-1];
    }

    //Clean up
    delete[] line;
}

/// ARFilter implementation


// Constructor
ARFilter::ARFilter(const std::vector<rsFloat> &coeffs)
{
    // Get the filter order
    order = coeffs.size();

    // Allocate memory to store co-efficients and state
    filter = new rsFloat[order];
    w = new rsFloat[order];

    // Load the co-efficients from the vectors into the arrays
    for (unsigned int i = 0; i < order; i++) {
        filter[i] = coeffs[i];
        w[i] = 0;
    }
}

// Destructor
ARFilter::~ARFilter()
{
    // Clean up memory
    delete[] filter;
    delete[] w;
}

/// Pass a single sample through the filter
rsFloat ARFilter::Filter(rsFloat sample)
{
    // Shift the filter state
    for (unsigned int j = order-1; j > 0; j--)
        w[j] = w[j-1];

    // Calculate w[0]
    w[0] = sample;
    for (unsigned int j = 1; j < order; j++)
        w[0] -= filter[j]*w[j];

    // Return the output value of the filter
    return w[0];
}

/// Pass an array of samples through the filter, filtering in place
void ARFilter::Filter(rsFloat *samples, int size)
{
    for (int i = 0; i < size; i++)
    {
        //Shift the filter state
        for (unsigned int j = order-1; j > 0; j--)
            w[j] = w[j-1];

        // Calculate w[0]
        w[0] = samples[i];
        for (unsigned int j = 1; j < order; j++)
            w[0] -= filter[j]*w[j];

        //Calculate y[n]
        samples[i] = w[0];
    }
}

/// Upsampler implementation

// Constructor
Upsampler::Upsampler(int ratio):
    ratio(ratio)
{
    // Create the FIR interpolation filter
    filter_size = 8*ratio+1; // 8*ratio should give adequate performance

    // Allocate memory for the filter bank
    filterbank = new rsFloat[filter_size];

    // Simple windowed sinc filter design procedure; Hamming window provides a solid tradeoff between rolloff and stopband attenuation
    for (int i = 0; i < filter_size; i++) {
        rsFloat window_value = 0.54 - 0.46 * std::cos(2*M_PI*i/(rsFloat)(filter_size));
        rsFloat filter_value = Sinc(1.0/(rsFloat)(ratio)*(i-filter_size/2));
        filterbank[i] = filter_value * window_value;
    }

    // Allocate memory for the sample state
    sample_memory = new rsFloat[filter_size/ratio+1];

    // Clear sample memory to zero
    for (int i = 0; i < filter_size/ratio+1; i++)
    sample_memory[i] = 0;
}

// Destructor
Upsampler::~Upsampler()
{
    // Clean up filter and state
    delete[] filterbank;
    delete[] sample_memory;
}

// Get a sample, from either the provided pointer or sample memory
inline rsFloat Upsampler::GetSample(rsFloat *samples, int n)
{
    if (n >= 0)
        return samples[n];
    else
        return sample_memory[n+filter_size];
}

/// Upsamples a signal and applies an anti-imaging filter
void Upsampler::Upsample(rsFloat *insamples, int in_size, rsFloat *outsamples, int out_size)
{
    // Check the target array size
    if (out_size != (ratio*in_size))
        throw std::runtime_error("Target array size is not correct in Upsample");

    // Polyphase upsampler implementation
    // Follows the diagram in section 4.7.4 "Polyphase Implementation of Interpolation Filters" of
    // Discrete Time Signal Processing, 2nd ed., Oppenheim and Schafer
    for (int i = 0, branch = 0; i < in_size; i++, branch++)
    {
        if (branch >= ratio)
            branch = 0;
        outsamples[i] = 0;

        // Apply the branch filter to the data
        for (int j = branch; j < filter_size; j += ratio)
            outsamples[i] += filterbank[j] * GetSample(insamples, i-j/ratio);
    }

    // Update the sample history
    int transfer_size = filter_size/ratio+1;
    if (in_size >= transfer_size)
        memcpy(sample_memory, &(insamples[in_size-transfer_size]), transfer_size*sizeof(rsFloat));
    else {
        // Shift existing samples
        for (int i = 0; i < (transfer_size-in_size); i++)
            sample_memory[i] = sample_memory[i+in_size];

        // Add new samples to the end of the buffer
        for (int i = 0; i < in_size; i++)
            sample_memory[i+transfer_size-in_size] = insamples[i];
    }
}

/// DecadeUpsample Implementation

// Constructor
DecadeUpsampler::DecadeUpsampler()
{
    // Denominator coefficients for 11th order elliptic lowpass at 0.1fs
    rsFloat den_coeffs[12] =
    {
        1.0,
        -10.301102119865,
        48.5214567642597,
        -137.934509572412,
        262.914952985445,
        -352.788381841481,
        340.027874008585,
        -235.39260470286,
        114.698499845697,
        -37.4634653062448,
        7.38208765922137,
        -0.664807695826097
    };

    // Numerator coefficients for 11th order elliptic lowpass at 0.1fs
    rsFloat num_coeffs[12] =
    {
        2.7301694322809e-06,
       -1.8508123430239e-05,
       5.75739466753894e-05,
       -0.000104348734423658,
       0.000111949190289715,
       -4.9384188225528e-05,
       -4.9384188225522e-05,
       0.00011194919028971,
       -0.000104348734423656,
       5.75739466753884e-05,
       -1.85081234302388e-05,
       2.73016943228086e-06
    };

    // Initialize the anti-imaging filter
    filter = new IIRFilter(den_coeffs, num_coeffs, 12);
}

// Destructor
DecadeUpsampler::~DecadeUpsampler() {
    delete filter;
}


/// Upsample a single sample at a time
void DecadeUpsampler::Upsample(rsFloat sample, rsFloat *out)
{
    // Prepare the output array
    out[0] = sample;
    for (int i = 1; i < 10; i++)
        out[i] = 0;

    // Filter in place
    filter->Filter(out, 10);
}

// Upsample a whole batch of samples
void DecadeUpsampler::Upsample(rsFloat *in, int count, rsFloat *out)
{
    // Prepare the array for filtering
    for (int i = 0; i < count; i++) {
        out[i*10] = in[i];
        for (int j = 1; j < 10; j++)
            out[i*10+j] = 0;
    }

    // Filter in place
    filter->Filter(out, count*10);
}
