#include "hip/hip_runtime.h"
//rspattern.cpp
// Implementation of interpolated 2D arrays for gain patterns and RCS patterns
// Marc Brooker mbrooker@rrsg.ee.uct.ac.za
//19 September 2007

#include <config.h>
#include <cmath>
#include "rspattern.cuh"
#include "rshdf5.cuh"

using namespace rs;

//
// Pattern Implementation
//

/// Constructor
Pattern::Pattern(const std::string &filename)
{
  //Load the pattern from the HDF5 file
  pattern = rshdf5::ReadPattern(filename, "antenna", size_azi, size_elev);
}

/// Destructor
Pattern::~Pattern()
{
  //Clean up
  for (unsigned int i = 0; i < size_azi; i++)
    delete[] pattern[i];
  delete[] pattern;
}

/// Get the gain at the given angle
rsFloat Pattern::GetGain(const rs::SVec3 &angle) const
{
  //Get the nearest points in elevation and azimuth
  double x1 = std::floor((angle.azimuth+M_PI)/(2*M_PI)*(size_azi-1))/(double)(size_azi-1);
  double ex1 = (angle.azimuth+M_PI)/(2*M_PI);
  double x2 = x1+1.0/(double)(size_azi);
  double y1 = std::floor((angle.elevation+M_PI)/(2*M_PI)*(size_elev-1))/(double)(size_elev-1);
  double ey1 = (angle.elevation+M_PI)/(2*M_PI);
  double y2 = y1+1.0/(double)(size_elev);
  //Get the interpolation constants
  double t = (ex1-x1)/(x2-x1);
  double u = (ey1-y1)/(y2-y1);

  //Get the offsets into the array
  int arr_x = std::floor(x1*size_azi);
  int arr_y = std::floor(y1*size_elev);

  //Get the interpolated value, using bilinear interpolation
  double interp = (1.0-t)*(1.0-u)*pattern[arr_x][arr_y];
  interp += t*(1.0-u)*pattern[(arr_x+1)%size_azi][arr_y];
  interp += t*u*pattern[(arr_x+1)%size_azi][(arr_y+1)%size_elev];
  interp += (1.0-t)*u*pattern[arr_x][(arr_y+1)%size_elev];
  return interp;
}
