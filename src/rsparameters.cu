/// rsparameters.cpp - Implementation of Singleton class to hold common simulation parameters
/// Marc Brooker, 11 June 2006
/// Edited by Yaaseen Martin, 27 August 2019

#include <stdexcept>
#include "time.h"
#include "rsparameters.cuh"
#include "rsdebug.cuh"

using namespace rs;

rsParameters* rsParameters::instance = new rsParameters();

namespace {
    struct SimParameters {
        rsFloat c; // Propagation speed of the wave in the medium
        rsFloat start; // The start time of the simulation
        rsFloat end; // The end time of the simulation
        rsFloat cw_sample_rate; // The number of samples per second to take of changes in the CW state
        rsFloat rate; // The sample rate to use for rendering
        unsigned int random_seed; // The seed used for random number calculations
        unsigned int adc_bits; // The number of bits to use for quantization
        unsigned int filter_length; // The length of the filter for rendering purposes
        rsParms::BinaryFileType filetype; // The type of binary files produced by binary rendering
        bool export_xml; // Export results in XML format
        bool export_csv; // Export results in CSV format
        bool export_binary; // Export results in binary format
        unsigned int render_threads; // Number of threads to use to render each receiver
        unsigned int oversample_ratio; // Ratio of oversampling applied to pulses before rendering
    };

    /// Object which contains all the simulation parameters
    SimParameters sim_parms;
}

// Private constructor for rsParameter, should only be called once
rsParameters::rsParameters() {

    // Default value of c, speed of light in a vacuum
    sim_parms.c = 299792458.0;

    // Simulation defaults to zero length
    sim_parms.start = 0;
    sim_parms.end = 0;

    // CW Interpolation rate defaults to 1000 per second
    sim_parms.cw_sample_rate = 1000;

    // Oversample by default
    sim_parms.rate = 0;

    // Default filter length is 33
    sim_parms.filter_length = 33;

    // Binary file type defaults to CSV
    sim_parms.filetype = rsParms::RS_FILE_SOARSBIN;

    // Don't export xml by default
    sim_parms.export_xml = false;

    // Don't export csv by default
    sim_parms.export_csv = false;

    // Export binary by default
    sim_parms.export_binary = true;

    // The random seed is set the to the current time by default
    sim_parms.random_seed = static_cast<unsigned int>(time(NULL));

    // The default is not to quantize
    sim_parms.adc_bits = 0;

    // Default maximum number of render threads
    sim_parms.render_threads = 1;

    // Default is to disable oversampling
    sim_parms.oversample_ratio = 1;
}

rsParameters *rsParameters::modify_parms()
{
    if (!instance)
        instance = new rsParameters;
    return instance;
}

// Getters for settings
rsFloat rsParameters::c()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.c;
}

// Return Boltzmann's constant, k
rsFloat rsParameters::boltzmann_k()
{
    return 1.3806503e-23;
}

rsFloat rsParameters::start_time()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.start;
}

rsFloat rsParameters::end_time()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.end;
}

rsFloat rsParameters::cw_sample_rate()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.cw_sample_rate;
}

rsParms::BinaryFileType rsParameters::binary_file_type()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.filetype;
}

rsFloat rsParameters::rate()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.rate;
}

unsigned int rsParameters::random_seed()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.random_seed;
}

unsigned int rsParameters::adc_bits()
{
    if (!instance)
        instance = new rsParameters();
    return sim_parms.adc_bits;
}

bool rsParameters::export_xml()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.export_xml;
}

bool rsParameters::export_csv()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.export_csv;
}

bool rsParameters::export_binary()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.export_binary;
}

/// Length to use for the rendering filter
unsigned int rsParameters::render_filter_length()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.filter_length;
}

/// Maximum number of threads to use for rendering
unsigned int rsParameters::render_threads() {
    if (!instance)
        instance = new rsParameters;
    return sim_parms.render_threads;
}

unsigned int rsParameters::oversample_ratio()
{
    if (!instance)
        instance = new rsParameters;
    return sim_parms.oversample_ratio;
}

/// Setters for global parameters

void rsParameters::SetC(rsFloat c)
{
    sim_parms.c = c;
}

void rsParameters::SetTime(rsFloat start, rsFloat end)
{
    sim_parms.start = start;
    sim_parms.end = end;
}

void rsParameters::SetCWSampleRate(rsFloat rate)
{
    sim_parms.cw_sample_rate = rate;
}

void rsParameters::SetRate(rsFloat factor)
{
    sim_parms.rate = factor;
}

void rsParameters::SetRandomSeed(unsigned int random_seed)
{
    sim_parms.random_seed = random_seed;
}

void rsParameters::SetBinaryFileType(rsParms::BinaryFileType type)
{
    sim_parms.filetype = type;
}

void rsParameters::SetExporters(bool xml, bool csv, bool binary)
{
    sim_parms.export_xml = xml;
    sim_parms.export_csv = csv;
    sim_parms.export_binary = binary;
}

void rsParameters::SetADCBits(unsigned int bits)
{
    sim_parms.adc_bits = bits;
}

void rsParameters::SetRenderFilterLength(unsigned int length)
{
    // Sanity check the render filter length
    if (length < 16)
        throw std::runtime_error("ERROR: Render filter length must be > 16");
    sim_parms.filter_length = length;
    rsDebug::printf(rsDebug::RS_VERY_VERBOSE, "Render filter length set to custom value: %d\n", length);
}

void rsParameters::SetOversampleRatio(unsigned int ratio)
{
    //Sanity check the ratio
    if (ratio == 0)
        throw std::runtime_error("ERROR: Oversample ratio must be >= 1");
    sim_parms.oversample_ratio = ratio;
}

void rsParameters::SetThreads(unsigned int threads)
{
    sim_parms.render_threads = threads;
}
